#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdio.h>
#include <math.h>       /* ceil */
#include <map>
#include <vector>

// Max Threads per block in GeForce 210
#define TxB 512

__global__
void image_equalization_kernel(const uchar4* const rgbaImage,
                                     uchar4* const outputImage,
                                     int numRows, int numCols)
{
  // El mapeo de los componentes uchar4 aRGBA es:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  //Nota: Ignoramos el canal alfa
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if( i < numRows*numCols){
    uchar4 px = rgbaImage[i]; // thread pixel to process
    //printf( "Antes: R: %i G: %i B %i \n",px.x,px.y,px.z );


    __shared__ unsigned int channelR[256];
    __shared__ unsigned int channelG[256];
    __shared__ unsigned int channelB[256];

    __shared__ unsigned int channelRacc[256];
    __shared__ unsigned int channelGacc[256];
    __shared__ unsigned int channelBacc[256];

    __shared__ unsigned char channelReq[256];
    __shared__ unsigned char channelGeq[256];
    __shared__ unsigned char channelBeq[256];


    if( i == 0 ){
      for(int q = 0; q < 256; q++ ){
        channelR[q]    = 0;
        channelG[q]    = 0;
        channelB[q]    = 0;
        channelRacc[q] = 0;
        channelGacc[q] = 0;
        channelBacc[q] = 0;
      }
    }

    __syncthreads();

    //Calcular histograma
    atomicAdd( &channelR[ px.x ] , 1);
    atomicAdd( &channelG[ px.y ] , 1);
    atomicAdd( &channelB[ px.z ] , 1);

    __syncthreads();

    if (i == 0){

      //Impresión del histograma
      // printf("Canal R \n");
      // for(int k = 0; k < 256; k++)
      //   printf("%i - %i \n", k, channelR[k]);
      
      // printf("Canal G \n");
      // for(int k = 0; k < 256; k++)
      //   printf("%i - %i \n", k, channelG[k]);

      // printf("Canal B \n");
      // for(int k = 0; k < 256; k++)
      //   printf("%i - %i \n", k, channelB[k]);        

      //Sumas acumuladas
      channelRacc[0] = channelR[0];
      channelGacc[0] = channelG[0];
      channelBacc[0] = channelB[0];
      for( int j = 1; j < 256; j++ ){
        channelRacc[j] = channelRacc[j-1] + channelR[j];
        channelGacc[j] = channelGacc[j-1] + channelG[j];
        channelBacc[j] = channelBacc[j-1] + channelB[j];
      }

      // printf("Canal R acc \n");
      // for(int k = 0; k < 256; k++)
      //   printf("%i - %i \n", k, channelRacc[k]);

    }

    __syncthreads();
    //Ecualizamos con los primeros 256 hilos.
    if(i < 256){

      channelReq[i] = round((float)channelRacc[i] * 255.0/( (float)numRows*numCols ) );
      channelGeq[i] = round((float)channelGacc[i] * 255.0/( (float)numRows*numCols ) );
      channelBeq[i] = round((float)channelBacc[i] * 255.0/( (float)numRows*numCols ) );

    }
    
    __syncthreads();

    outputImage[i].x =  channelReq[px.x];
    outputImage[i].y =  channelGeq[px.y];
    outputImage[i].z =  channelBeq[px.z];

    __syncthreads();
    //printf( "Despues: R: %i G: %i B %i \n",outputImage[i].x,outputImage[i].y,outputImage[i].z );
  }
  
}

void image_equalization(uchar4 * const d_rgbaImage,
                  uchar4* const d_outputImage, size_t numRows, size_t numCols)
{

  // Dado que no importa la posicion relativa de los pixels
  // en este algoritmo, la estrategia para asignar hilos a
  // bloques y rejillas sera sencillamente la de cubrir
  // a todos los pixeles con hebras en el eje X
  long long int total_px = numRows * numCols;  // total pixels
  long int grids_n = ceil(total_px / TxB); // grids numer
  const dim3 blockSize(TxB, 1, 1);
  const dim3 gridSize(grids_n, 1, 1);
  image_equalization_kernel<<<gridSize, blockSize>>>(d_rgbaImage, d_outputImage, numRows, numCols);
  
  hipDeviceSynchronize(); 
  checkCudaErrors(hipGetLastError());
}