#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdio.h>
#include <math.h>       /* ceil */

// Max Threads per block in GeForce 210
#define TxB 512

//Kernel corrección gamma
__global__
void gamma_correction_kernel(const uchar4* const rgbaImage,
                                   uchar4* const outputImage,
                             int numRows, int numCols, float gamma)
{
  // El mapeo de los componentes uchar4 aRGBA es:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  //Nota: Ignoramos el canal alfa
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if( i < numRows*numCols){
    uchar4 px = rgbaImage[i]; // thread pixel to process
    //printf( "Antes: R: %i G: %i B %i \n",px.x,px.y,px.z );

    unsigned char rcolor = round(pow((px.x / 255.0f), (1.0f / gamma)  ) * 255.0f);
    outputImage[i].x = (rcolor > 255.0f) ? 255.0f : rcolor;
    unsigned char gcolor = round(pow((px.y / 255.0f), (1.0f / gamma)  ) * 255.0f);
    outputImage[i].y = (gcolor > 255.0f) ? 255.0f : gcolor;
    unsigned char bcolor = round(pow((px.z / 255.0f), (1.0f / gamma)  ) * 255.0f);
    outputImage[i].z = (bcolor > 255.0f) ? 255.0f : bcolor;

    //printf( "Despues: R: %i G: %i B %i \n",outputImage[i].x,outputImage[i].y,outputImage[i].z );
  }
  
}

void gamma_correction(uchar4* const d_rgbaImage,
                      uchar4* const d_outputImage, 
                      size_t numRows, size_t numCols, float gamma)
{

  // Dado que no importa la posicion relativa de los pixels
  // en este algoritmo, la estrategia para asignar hilos a
  // bloques y rejillas sera sencillamente la de cubrir
  // a todos los pixeles con hebras en el eje X
  long long int total_px = numRows * numCols;  // total pixels
  long int grids_n = ceil(total_px / TxB); // grids numer
  const dim3 blockSize(TxB, 1, 1);
  const dim3 gridSize(grids_n, 1, 1);
  gamma_correction_kernel<<<gridSize, blockSize>>>(d_rgbaImage, d_outputImage, numRows, numCols, gamma);
  
  hipDeviceSynchronize(); 
  checkCudaErrors(hipGetLastError());
}