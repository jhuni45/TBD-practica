#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdio.h>
#include <math.h>       /* ceil */

// Max Threads per block in GeForce 210
#define TxB 512


//Kernel binarización de imagen
__global__
void image_binarization_kernel(const uchar4* const rgbaImage,
                               unsigned char* const outputImage,
                               int numRows, int numCols, int threshold)
{
  // El mapeo de los componentes uchar4 aRGBA es:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  //Nota: Ignoramos el canal alfa
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if( i < numRows*numCols){
    uchar4 px = rgbaImage[i]; // thread pixel to process
    //printf( "Antes: R: %i G: %i B %i \n",px.x,px.y,px.z );
   
    //Primero se convierte a escala de grises

    unsigned char grey = .299f * px.x +
                         .587f * px.y +
                         .114f * px.z;

    //Binarizamos (si gris > threshold ? 255 : 0)

    outputImage[i] = ( grey > threshold) ? 255 : 0;

  }
  
}

void image_binarization(uchar4* const d_rgbaImage, 
                        unsigned char* const d_outputImage, 
                        size_t numRows, size_t numCols, int threshold)
{

  // Dado que no importa la posicion relativa de los pixels
  // en este algoritmo, la estrategia para asignar hilos a
  // bloques y rejillas sera sencillamente la de cubrir
  // a todos los pixeles con hebras en el eje X
  long long int total_px = numRows * numCols;  // total pixels
  long int grids_n = ceil(total_px / TxB); // grids numer
  const dim3 blockSize(TxB, 1, 1);
  const dim3 gridSize(grids_n, 1, 1);
  image_binarization_kernel<<<gridSize, blockSize>>>(d_rgbaImage, d_outputImage, numRows, numCols, threshold);
  
  hipDeviceSynchronize(); 
  checkCudaErrors(hipGetLastError());
}