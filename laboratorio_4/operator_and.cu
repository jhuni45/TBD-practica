#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdio.h>
#include <math.h>       /* ceil */

// Max Threads per block in GeForce 210
#define TxB 512


//Kernel operador not
__global__
void operator_and_kernel(const unsigned char* const inputImage, 
                         const unsigned char* const inputImage2,
                               unsigned char* const outputImage,
                               int numRows, int numCols)
{
  // El mapeo de los componentes uchar4 aRGBA es:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  //Nota: Ignoramos el canal alfa
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if( i < numRows*numCols){
   
    if( (inputImage[i] == 255) && (inputImage2[i] == 255)  )
      outputImage[i] = 255;
    else
      outputImage[i] = 0;
    //Cambiamos si es 0:255 , si es 255:0
    //outputImage[i] = ( (inputImage[i] == 255) && (inputImage2[i] == 255) ) ? 255 : 0;
    //printf("%i and %i = %i  \n",inputImage[i], inputImage2[i], outputImage[i]);

  }
  
}

void operator_and_image(unsigned char* const d_inputImage, 
                        unsigned char* const d_inputImage2, 
                        unsigned char* const d_outputImage, 
                        size_t numRows, size_t numCols)
{

  // Dado que no importa la posicion relativa de los pixels
  // en este algoritmo, la estrategia para asignar hilos a
  // bloques y rejillas sera sencillamente la de cubrir
  // a todos los pixeles con hebras en el eje X
  long long int total_px = numRows * numCols;  // total pixels
  long int grids_n = ceil(total_px / TxB); // grids numer
  const dim3 blockSize(TxB, 1, 1);
  const dim3 gridSize(grids_n, 1, 1);
  operator_and_kernel<<<gridSize, blockSize>>>(d_inputImage, d_inputImage2, d_outputImage, numRows, numCols);
  
  hipDeviceSynchronize(); 
  checkCudaErrors(hipGetLastError());
}