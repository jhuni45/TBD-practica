#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdio.h>
#include <math.h>       /* ceil */

// Max Threads per block in GeForce 210
#define TxB 512


//Kernel operador not
__global__
void operator_not_kernel(const unsigned char* const inputImage,
                               unsigned char* const outputImage,
                               int numRows, int numCols)
{
  // El mapeo de los componentes uchar4 aRGBA es:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  //Nota: Ignoramos el canal alfa
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if( i < numRows*numCols){
   
    //Cambiamos si es 0:255 , si es 255:0

    outputImage[i] = ( inputImage[i] == 0 ) ? 255 : 0;

  }
  
}

void operator_not_image(unsigned char* const d_inputImage, 
                  unsigned char* const d_outputImage, 
                  size_t numRows, size_t numCols)
{

  // Dado que no importa la posicion relativa de los pixels
  // en este algoritmo, la estrategia para asignar hilos a
  // bloques y rejillas sera sencillamente la de cubrir
  // a todos los pixeles con hebras en el eje X
  long long int total_px = numRows * numCols;  // total pixels
  long int grids_n = ceil(total_px / TxB); // grids numer
  const dim3 blockSize(TxB, 1, 1);
  const dim3 gridSize(grids_n, 1, 1);
  operator_not_kernel<<<gridSize, blockSize>>>(d_inputImage, d_outputImage, numRows, numCols);
  
  hipDeviceSynchronize(); 
  checkCudaErrors(hipGetLastError());
}