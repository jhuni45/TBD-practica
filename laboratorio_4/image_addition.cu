#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdio.h>
#include <math.h>       /* ceil */

// Max Threads per block in GeForce 210
#define TxB 512


//Kernel suma media ponderada
__global__
void image_addition_kernel(const uchar4* const rgbaImage,
                           const uchar4* const rgbaImage2,
                                 uchar4* const outputImage,
                           int numRows, int numCols, float percent)
{
  // El mapeo de los componentes uchar4 aRGBA es:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  //Nota: Ignoramos el canal alfa
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if( i < numRows*numCols){
    uchar4 px = rgbaImage[i]; // thread pixel to process
    uchar4 px2 = rgbaImage2[i]; // thread pixel to process
    //printf( "Antes: R: %i G: %i B %i \n",px.x,px.y,px.z );

    outputImage[i].x = ( percent * px.x ) + ( ( 1 - percent ) * px2.x );
    outputImage[i].y = ( percent * px.y ) + ( ( 1 - percent ) * px2.y );
    outputImage[i].z = ( percent * px.z ) + ( ( 1 - percent ) * px2.z );
    

    //printf( "Despues: R: %i G: %i B %i \n",outputImage[i].x,outputImage[i].y,outputImage[i].z );
  }
  
}

void image_addition(uchar4* const d_rgbaImage, 
                    uchar4* const d_rgbaImage2,
                    uchar4* const d_outputImage, 
                    size_t numRows, size_t numCols, float percent)
{

  // Dado que no importa la posicion relativa de los pixels
  // en este algoritmo, la estrategia para asignar hilos a
  // bloques y rejillas sera sencillamente la de cubrir
  // a todos los pixeles con hebras en el eje X
  long long int total_px = numRows * numCols;  // total pixels
  long int grids_n = ceil(total_px / TxB); // grids numer
  const dim3 blockSize(TxB, 1, 1);
  const dim3 gridSize(grids_n, 1, 1);
  image_addition_kernel<<<gridSize, blockSize>>>(d_rgbaImage, d_rgbaImage2, d_outputImage, numRows, numCols, percent);
  
  hipDeviceSynchronize(); 
  checkCudaErrors(hipGetLastError());
}