#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cstdio>
#include <iostream>
#include <opencv2/opencv.hpp>
#include "convolucion.h"
#include <hip/hip_runtime.h>
#include <string>
#include <math.h>
#define TxB 512

using namespace std;
using namespace cv;


Mat imageInput;
Mat imageOutput;

uchar4 *d_inputImg__;
unsigned char *d_outputImg__;

float *h_filter;

size_t numRows() { return imageInput.rows; }
size_t numCols() { return imageInput.cols; }

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    exit(1);
  }
}

void preProcess(uchar4 **imgInput, unsigned char **imgOutput, uchar4 **d_inputImg, unsigned char **d_outputImg, string& nameImg){
      // revisar que no halla errores
      checkCudaErrors(hipFree(0));

      Mat image;
      image = imread(nameImg.c_str(),CV_LOAD_IMAGE_COLOR);
      if (image.empty()) {
            cout << "No se puede abrir el archivo: " <<nameImg <<endl;
            exit(1);
      }
      cvtColor(image, imageInput, CV_BGR2RGBA);

      imageOutput.create(image.rows, image.cols, CV_8UC1);

      if (!imageInput.isContinuous() || !imageOutput.isContinuous()) {
            std::cerr << "No son imagenes continuas" << std::endl;
            exit(1);
          }

      // dando tamaño a los arrays inputImagen y imagenBlur que se pasaran a gpu
      *imgInput = (uchar4*)imageInput.ptr<unsigned char>(0);
      *imgOutput = imageOutput.ptr<unsigned char>(0);

      const size_t numPixels = numCols()* numRows();

      checkCudaErrors(hipMalloc(d_inputImg, sizeof(uchar4) * numPixels));
      checkCudaErrors(hipMalloc(d_outputImg, sizeof(unsigned char) * numPixels));
      checkCudaErrors(hipMemset(*d_outputImg, 0, numPixels * sizeof(unsigned char)));

      checkCudaErrors(hipMemcpy(*d_inputImg, *imgInput, sizeof(uchar4) * numPixels, hipMemcpyHostToDevice));

      d_inputImg__ = *d_inputImg;
      d_outputImg__ = *d_outputImg;
}

// __global__ blurKernel()


__global__
void rgba_to_grey_kernel(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
  // The mapping from components of a uchar4 to RGBA is:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  //
  //The output (greyImage) at each pixel should be the result of
  //applying the formula: output = .299f * R + .587f * G + .114f * B;
  //Note: We will be ignoring the alpha channel for this conversion
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  uchar4 px = rgbaImage[i]; // thread pixel to process
  greyImage[i] = .299f * px.x +
                 .587f * px.y +
                 .114f * px.z;
}

void rgba_to_grey(uchar4 * const d_rgbaImage,
                  unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{

  // Since it does not matter the relative position of a pixel
  // the block - grid assign strategy will simply be to cover
  // all pixels secuencially in 'x' axis
  long long int total_px = numRows * numCols;  // total pixels
  long int grids_n = ceil(total_px / TxB); // grids numer
  const dim3 blockSize(TxB, 1, 1);
  const dim3 gridSize(grids_n, 1, 1);
  rgba_to_grey_kernel<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}

__global__ void convolutionGPU(float *d_Result, float *d_Data, int dataW, int dataH )
{
      
      // global mem address for this thread
      const int gLoc = threadIdx.x +  blockIdx.x * blockDim.x + threadIdx.y * dataW + blockIdx.y * blockDim.y * dataW; 

      float sum = 0;
      float value = 0;
      
      for (int i = -KERNEL_RADIUS; i <= KERNEL_RADIUS; i++)	// row wise
      for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++)	// col wise
      {
            // check row first
            if (blockIdx.x == 0 && (threadIdx.x + i) < 0)	// left apron
                  value = 0;
            else if ( blockIdx.x == (gridDim.x - 1) && (threadIdx.x + i) > blockDim.x-1 )	// right apron
            value = 0;
      else { 
            // check col next
            if (blockIdx.y == 0 && (threadIdx.y + j) < 0)	// top apron
            value = 0;
            else if ( blockIdx.y == (gridDim.y - 1) && 
            (threadIdx.y + j) > blockDim.y-1 )	// bottom apron
            value = 0;
            else	// safe case
            value = d_Data[gLoc + i + j * dataW];
      } 
      sum += value * d_Kernel[KERNEL_RADIUS + i] * d_Kernel[KERNEL_RADIUS + j];
      }
      d_Result[gLoc] = sum; 
}


void convolutionGPU(float *d_Result, float *d_Data, int dataW, int dataH )
{
      
      long long int total_px = dataH * dataW;  // total pixels
      long int grids_n = ceil(total_px / TxB); // grids numer
      const dim3 blockSize(TxB, 1, 1);
      const dim3 gridSize(grids_n, 1, 1);
      rgba_to_grey_kernel<<<gridSize, blockSize>>>(d_Data, d_Result, dataW, dataH);
      
      hipDeviceSynchronize(); checkCudaErrors(hipGetLastError()); 
}


int main(int argc, char **argv) {
      uchar4        *h_inputImage, *d_inputImage;
      unsigned char *h_outputImage, *d_outputImage;
    
      string nameImage;
      string outNameImage;

      // creando memmoria de 3x3 para el filtro en host
      float *h_filter;
      float *d_filter;
      unsigned int h_ tam_filter = 3;
      unsigned int d_tam_filter = h_tam_filter*h_tam_filter*sizeof(float);

      h_filter = new float[h_tam_filter*h_tam_filter];
    
      //Evaluando errores
      checkCudaErrors(hipFree(0));
    
      switch (argc)
      {
          case 2:
            nameImage = std::string(argv[1]);
            outNameImage= "output.png";
            break;
          default:
          std::cerr << "Use: ./cuda [nombre_imagen]" << std::endl;
          exit(1);
      }
      // Cargando la imagen y devolviendonos los punteros de entra y salida
      preProcess(&h_inputImage, &h_outputImage, &d_inputImage, &d_outputImage, nameImage);


      //alocando espacio de memoria para el filtro en device
      hipMalloc(d_filter,d_tam_filter);

    
      //llamando a la funcion Cuda
      // rgba_to_grey(d_inputImage, d_outputImage, numRows(), numCols());
      convolutionGPU(d_outputImage, d_inputImage, numRows(), numCols())
    
      size_t numPixels = numRows()*numCols();
      checkCudaErrors(hipMemcpy(h_outputImage, d_outputImage, sizeof(unsigned char) * numPixels, hipMemcpyDeviceToHost));
    
      /* Output the grey image */
      cv::Mat outputImg(numRows(), numCols(), CV_8UC1, (void*)h_outputImage);
      // Open the window
      cv::namedWindow("Nueva Imagen");
      // Display the image m in this window
      cv::imshow("Nueva Imagen", outputImg);
      cvWaitKey (0);
      cvDestroyWindow ("Nueva Imagen");
      //output the image
      cv::imwrite(outNameImage.c_str(), outputImg);
    
      /* Cleanup */
      hipFree(d_inputImg__);
      hipFree(d_outputImg__);
    
      return 0;
    }