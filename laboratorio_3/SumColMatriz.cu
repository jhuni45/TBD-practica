#include "hip/hip_runtime.h"
%%cu
#include <iostream>
using namespace std;

__global__ void SumaColMatrizKernel_0(int M, int N, float* Md, float* Nd){
    // Pvalue es usado para el valor intermedio
    int Pvalue = 0;
    int columna = blockIdx.x * blockDim.x + threadIdx.x;
    int posIni = columna * M;
        for (int k = 0; k < M; ++k) {
            Pvalue = Pvalue + Md[posIni + k];
        }
    Nd[columna] = Pvalue;
}

__global__ void SumaColMatrizKernel_1(int M, float* Md, float* Nd)
{
    // Pvalue es usado para el valor intermedio
    int Pvalue = 0;
    int columna = threadIdx.x;
    int posIni = columna * M;
    for (int k = 0; k < M; ++k) {
        for (int k =
            0; k < M; ++k) {
            Pvalue = Pvalue + Md[posIni + k];
        }
        Nd[columna] = Pvalue;
    }
}

__global__ void SumaColMatrizKernel_2(int M, int N, float* Md, float* Nd)
{
    // Pvalue es usado para el valor intermedio
    int Pvalue = 0;
    int columna = blockIdx.x * blockDim.x + threadIdx.x;
    int posIni = columna * M;
        for (int k = 0; k < M; ++k) {
            Pvalue = Pvalue + Md[posIni + k];
        }
    Nd[columna] = Pvalue;
}

__global__ void SumaColMatrizKernel_3(int M, float* Md, float* Nd)
{
    // Pvalue es usado para el valor intermedio
    int Pvalue = 0;
    int columna = blockIdx.x;
    int pasos = M / blockDim.x;
    int posIni = columna * M + threadIdx.x * pasos;
    for (int k = 0; k < pasos; ++k) {
        Pvalue = Pvalue + Md[posIni + k];
    }
    atomicAdd(&(Nd[columna]), Pvalue);
}

// Lanzamiento del kernel 4 con memoria bidimensional
__global__ void SumaColMatrizKernel_4(int M, int N, float* Md, float* Nd)
{
    // Pvalue es usado para el valor intermedio
    int Pvalue = 0;
    int columna = blockIdx.y * (N / gridDim.x) + threadIdx.x;
    int pasos = M / blockDim.x;
    int posIni = columna * M + threadIdx.x * pasos;
    for (int k = 0; k < pasos; ++k) {
        Pvalue = Pvalue + Md[posIni + k];
    }
    atomicAdd(&(Nd[columna]), Pvalue);
}

// Lanzamiento del kernel 5 con memoria compartida  
/*#define DIMBLOCKX 32
__global__ void SumaColMatrizKernel_5(int M, float* Md, float* Nd)
{
    __shared__ float Nds[DIMBLOCKX];
    int Pvalue = 0;
    int columna = blockIdx.x;
    int pasos = M / blockDim.x;
    int posIni = columna * M + threadIdx.x * pasos;
    for (int k = 0; k < pasos; ++k) {
        Pvalue = Pvalue + Md[posIni + k];
    }
    atomicAdd(&(Nd[columna]), Pvalue);
    // Nds[threadIdx.x] = Pvalue;

    __syncthreads();
    if (threadIdx.x == 0)
    {
        for (int i = 1; i < blockDim.x; ++i) {
            Nds[0] = Nds[0] + Nds[i];
        }
        atomicAdd(&(Nd[blockIdx.y]), Nds[0]);
        // Nd[blockIdx.y] = Nds[0];
    }
}*/

// Lanzamiento del kernel 6 con memoria compartida y memoria bidimensional
__global__ void SumaColMatrizKernel_6(int M, float* Md, float* Nd){
    // Pvalue es usado para el valor intermedio
    float Pvalue = 0;
    int columna = threadIdx.x;
    int posIni = columna*M;
    for (int k = 0; k < M; ++k) {
      Pvalue = Pvalue + Md[posIni+k];
    }
    Nd[columna] = Pvalue;
  }

void SumaColMatriz(int M, int N, float * Mh, float*  Nh){
    int size = M * N * sizeof(float), size2 = N*sizeof(float);
    float* Md, *Nd;

    // Asignar en dispositivo
    hipMalloc(&Md, size);
    hipMalloc(&Nd, size2);

    // Inicializo matrices en el dispositivo
    hipMemcpy(Md, Mh, size, hipMemcpyHostToDevice);
    hipMemset(Nd,0, size2);

    /*--------- KERNEL 0 ---------*/
    // SumaColMatrizKernel_0 <<<N, 1 >>> (M, Md, Nd);

    /*--------- KERNEL 1 ---------*/
    // SumaColMatrizKernel_1 <<<1, N >>> (M, Md, Nd);
    // dim3 tamGrid(1, 1); //Grid dimensión
    // dim3 tamBlock(N, 1, 1); //Block dimensión
    // SumaColMatrizKernel_1 <<<1, N >>> (M, Md, Nd);

    /*--------- KERNEL 2 ---------*/
    // int bloques = N / 128; // se asume N múltiplo de 128
    // dim3 tamGrid(bloques, 1); //Grid dimensión
    // dim3 tamBlock(128, 1, 1); //Block dimensión
    // SumaColMatrizKernel_2 <<<bloques, 128 >>> (M, Md, Nd);

    /*--------- KERNEL 3 ---------*/
    // int chunk = 32; // Se asume M múltiplo de 32
    // dim3 tamGrid(N / chunk, 1); //Grid dimensión
    // dim3 tamBlock(M / chunk, chunk, 1); //Block dimensión
    // SumaColMatrizKernel_3 <<<N, M / chunk >>> (M, Md, Nd);

    // Lanzamiento del kernel 4 con memoria bidimensional
    /*--------- KERNEL 4 ---------*/
    /* Si quiero que un bloque procese más de una columna */
    // int chunk = 32; // Se asume M y N múltiplos de 32
    // dim3 tamGrid(N / chunk, 1); //Grid dimensión
    // dim3 tamBlock(M / chunk, chunk, 1); //Block dimensión
    // SumaColMatrizKernel_4 <<<tamGrid, tamBlock >>> (M, N, Md, Nd);

    // Lanzamiento del kernel 5 con memoria compartida
    /*--------- KERNEL 5 ---------*/
    /* configuración de la ejecución */
    // int chunk = 32;
    // dim3 tamGrid(N, 1); //Grid dimensión
    // dim3 tamBlock(M / chunk, 1, 1); //Block dimensión
    // SumaColMatrizKernel_5 <<<tamGrid, tamBlock >>> (M, Md, Nd); /* lanzamiento del kernel */

    // Lanzamiento del kernel 6 con memoria compartida y memoria bidimensional
    /*--------- KERNEL 6 ---------*/
    /* configuración de la ejecución */
    dim3 tamGrid(1,1);
    dim3 tamBlock(N,1,1);
    SumaColMatrizKernel_6 <<<tamGrid, tamBlock>>>(M, Md, Nd);

    /*--------- TRAER RESULTADO ---------*/
    // Traer resultado;
    hipMemcpy(Nh, Nd, size2, hipMemcpyDeviceToHost);
    for (int i=0; i<N; i++)
      std::cout<<Nh[i]<<" ";

    // Free matrices en device
    hipFree(Md); // Free matrices en device
    hipFree(Nd); // Free matrices en device
}

int main(){
    int M =1024;
    int N =512;
    float *Mh = new float[M*N];
    float *Nh = new float[N];
    for (int i=0; i<M; i++)
      for (int j=0; j<N; j++)
        Mh[i*N+j] = 1;

    cout<<"Respuesta: "<<endl;
    SumaColMatriz(M,N,Mh,Nh);
}
