#include "hip/hip_runtime.h"
#include <assert.h> 
#include <hip/hip_runtime.h> 
#include <helper_functions.h> 
#include <hip/hip_runtime_api.h> 
#include <hip/device_functions.h>
#include <>
#include <stdio.h> 

__global__ void SumaColMatrizKernel_0(int M, int N, float* Md, float* Nd){
    // Pvalue es usado para el valor intermedio
    int Pvalue = 0;
    int columna = blockIdx.x * blockDim.x + threadIdx.x;
    int posIni = columna * M;
        for (int k = 0; k < M; ++k) {
            Pvalue = Pvalue + Md[posIni + k];
        }
    Nd[columna] = Pvalue;
}

__global__ void SumaColMatrizKernel_1(int M, float* Md, float* Nd)
{
    // Pvalue es usado para el valor intermedio
    int Pvalue = 0;
    int columna = threadIdx.x;
    int posIni = columna * M;
    for (int k = 0; k < M; ++k) {
        for (int k =
            0; k < M; ++k) {
            Pvalue = Pvalue + Md[posIni + k];
        }
        Nd[columna] = Pvalue;
    }
}

__global__ void SumaColMatrizKernel_2(int M, int N, float* Md, float* Nd)
{
    // Pvalue es usado para el valor intermedio
    int Pvalue = 0;
    int columna = blockIdx.x * blockDim.x + threadIdx.x;
    int posIni = columna * M;
        for (int k = 0; k < M; ++k) {
            Pvalue = Pvalue + Md[posIni + k];
        }
    Nd[columna] = Pvalue;
}

__global__ void SumaColMatrizKernel_3(int M, float* Md, float* Nd)
{
    // Pvalue es usado para el valor intermedio
    int Pvalue = 0;
    int columna = blockIdx.x;
    int pasos = M / blockDim.x;
    int posIni = columna * M + threadIdx.x * pasos;
    for (int k = 0; k < pasos; ++k) {
        Pvalue = Pvalue + Md[posIni + k];
    }
    atomicAdd(&(Nd[columna]), Pvalue);
}

__global__ void SumaColMatrizKernel_4(int M, int N, float* Md, float* Nd)
{
    // Pvalue es usado para el valor intermedio
    int Pvalue = 0;
    int columna = blockIdx.y * (N / gridDim.x) + threadIdx.x;
    int pasos = M / blockDim.x;
    int posIni = columna * M + threadIdx.x * pasos;
    for (int k = 0; k < pasos; ++k) {
        Pvalue = Pvalue + Md[posIni + k];
    }
    atomicAdd(&(Nd[columna]), Pvalue);
}

    // Lanzamiento del kernel 5 con memoria compartida  
#define DIMBLOCKX 32

__global__ void SumaColMatrizKernel_5(int M, float* Md, float* Nd)
{
    __shared__ float Nds[DIMBLOCKX];
    int Pvalue = 0;
    int columna = blockIdx.x;
    int pasos = M / blockDim.x;
    int posIni = columna * M + threadIdx.x * pasos;
    for (int k = 0; k < pasos; ++k) {
        Pvalue = Pvalue + Md[posIni + k];
    }
    atomicAdd(&(Nd[columna]), Pvalue);
    // Nds[threadIdx.x] = Pvalue;

    __syncthreads();
    if (threadIdx.x == 0)
    {
        for (int i = 1; i < blockDim.x; ++i) {
            Nds[0] = Nds[0] + Nds[i];
        }
        atomicAdd(&(Nd[blockIdx.y]), Nds[0]);
        // Nd[blockIdx.y] = Nds[0];
    }
}


void SumaColMatriz(int M, int N, float* Mh, float* Nh)
{
    int size = M * N * sizeof(int);
    int size2 = N * sizeof(float);
    float* Md, * Nd;

    // Asignar en dispositivo
    hipMalloc(&Md, size);
    hipMalloc(&Nd, size2);

    // Inicializo matrices en el dispositivo
    hipMemcpy(Md, Mh, size, hipMemcpyHostToDevice);
    hipMemset(Nd, 0, size2);
    // Invocar el kernel que suma en GPU

    /*--------- KERNEL 0 ---------*/
    // SumaColMatrizKernel_0 << <N, 1 >> > (M, Md, Nd);

    /*--------- KERNEL 1 ---------*/
    // SumaColMatrizKernel_1 << <1, N >> > (M, Md, Nd);
    // dim3 tamGrid(1, 1); //Grid dimensi�n
    // dim3 tamBlock(N, 1, 1); //Block dimensi�n
    // SumaColMatrizKernel_1 << <1, N >> > (M, Md, Nd);

    /*--------- KERNEL 2 ---------*/
    // int bloques = N / 128; // se asume N m�ltiplo de 128
    // dim3 tamGrid(bloques, 1); //Grid dimensi�n
    // dim3 tamBlock(128, 1, 1); //Block dimensi�n
    // SumaColMatrizKernel_2 << <bloques, 128 >> > (M, Md, Nd);

    /*--------- KERNEL 3 ---------*/
    // int chunk = 32; // Se asume M m�ltiplo de 32
    // dim3 tamGrid(N / chunk, 1); //Grid dimensi�n
    // dim3 tamBlock(M / chunk, chunk, 1); //Block dimensi�n
    // SumaColMatrizKernel_3 << <N, M / chunk >> > (M, Md, Nd);

    /*--------- KERNEL 4 ---------*/
    /* Si quiero que un bloque procese m�s de una columna */
    // int chunk = 32; // Se asume M y N m�ltiplos de 32
    // dim3 tamGrid(N / chunk, 1); //Grid dimensi�n
    // dim3 tamBlock(M / chunk, chunk, 1); //Block dimensi�n
    // SumaColMatrizKernel_4 << <tamGrid, tamBlock >> > (M, N, Md, Nd);


    // Lanzamiento del kernel 5 con memoria compartida
    /*--------- KERNEL 5 ---------*/
    // configuraci�n de la ejecuci�n
    int chunk = 32;
    dim3 tamGrid(N, 1); //Grid dimensi�n
    dim3 tamBlock(M / chunk, 1, 1); //Block dimensi�n
    SumaColMatrizKernel_5 << <tamGrid, tamBlock >> > (M, Md, Nd); /* lanzamiento del kernel */

    /*--------- TRAER RESULTADO ---------*/
    // Traer resultado;
    hipMemcpy(Nh, Nd, size2, hipMemcpyDeviceToHost);
    // Free matrices en device
    hipFree(Md);
    hipFree(Nd);
}

int main(void){
    int M = 1024;
    int N = 512;
    float** matriz;
    matriz = new float* [M];
    // En la posicion cero, se colaca todo el vector
    matriz[0] = new float[M * N];

    for (int i = 1; i < M; i++) {
        matriz[i] = &matriz[0][N];
    }

    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            matriz[i][j] = 1;
        }
    }

    float* Nh;
    Nh = new float[N];

    SumaColMatriz(M, N, matriz[0], Nh);
    for (int j = 0; j < N; j++){
        printf("%f ", Nh[j]);
    }

    return 0;
}